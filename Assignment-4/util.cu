#include "hip/hip_runtime.h"
#include "util.cuh"

__device__
Point rotate_point(Point anchor, Point grid_pos, int rot, int query_rows){
  if (rot == 0){
    return Point{anchor.x + grid_pos.x, anchor.y - (query_rows - 1) + grid_pos.y};
  }
  float angle = (float)rot * M_PI/180;
  float x = (float)grid_pos.x * cos(angle) - ((float)(query_rows - 1 - grid_pos.y) * sin(angle));
  float y = -((float)grid_pos.x * sin(angle)) - ((float)(query_rows - 1 - grid_pos.y) * cos(angle));
  return Point{anchor.x + x, anchor.y + y};
}

__device__
ll get_value(ll* arr, int i, int j, int rows, int cols){
  if(i < 0 || i >= rows || j < 0 || j >= cols){
    return 0;
  }
  return arr[i*cols + j];
}
__device__
void get_value(int* arr, int i, int j, int k, int rows, int cols, float *px){
  if(i < 0 || i >= rows || j < 0 || j >= cols){
    return;
  }
  for (int ch = 0; ch < 3; ch ++) {
    px[ch] = arr[i * cols * 3 + j * 3 + ch];
  }
}

__device__
ll get_prefix_sum(const BB& bb, int rows, int cols, ll* ps_mat){
  ll ret = 0;
  ret += get_value(ps_mat, bb.y, bb.x + bb.w, rows, cols);
  ret -= get_value(ps_mat, bb.y, bb.x - 1, rows, cols);
  ret -= get_value(ps_mat, bb.y - bb.h - 1, bb.x + bb.w, rows, cols);
  ret += get_value(ps_mat, bb.y - bb.h - 1, bb.x - 1, rows, cols);
  return ret;
}

__device__
void bilinear_interpolate(Point p, int ch, int rows, int cols, int* data, float *data_px){
  
  int xl = floor(p.x);
  int yl = floor(p.y);
  int xh = ceil(p.x);
  int yh = ceil(p.y);
  float bl_val[3]{0, 0, 0}, br_val[3]{0, 0, 0}, tl_val[3]{0, 0, 0}, tr_val[3]{0, 0, 0};
  get_value(data, yl, xl, ch, rows, cols, bl_val);
  get_value(data, yl, xh, ch, rows, cols, br_val);
  get_value(data, yh, xl, ch, rows, cols, tl_val);
  get_value(data, yh, xh, ch, rows, cols, tr_val);
  float x_frac, y_frac, x_inv_frac, y_inv_frac;
  if(xl == xh) {
    if (yl == yh) {
      data_px[0] = bl_val[0];
      data_px[1] = bl_val[1];
      data_px[2] = bl_val[2];
      return;
    }
    y_frac = (p.y - yl)/(yh - yl);
    y_inv_frac = 1 - y_frac;
    data_px[0] = bl_val[0] * y_inv_frac + tl_val[0] * y_frac;
    data_px[1] = bl_val[1] * y_inv_frac + tl_val[1] * y_frac;
    data_px[2] = bl_val[2] * y_inv_frac + tl_val[2] * y_frac;
    return;
  }
  if(yl == yh) {
    x_frac = (p.x - xl)/(xh - xl);
    x_inv_frac = 1 - x_frac;
    data_px[0] = bl_val[0] * x_inv_frac + br_val[0] * x_frac;
    data_px[1] = bl_val[1] * x_inv_frac + br_val[1] * x_frac;
    data_px[2] = bl_val[2] * x_inv_frac + br_val[2] * x_frac;
    return;
  }
  x_frac = (p.x - xl)/(xh - xl);
  y_frac = (p.y - yl)/(yh - yl);
  
  x_inv_frac = 1 - x_frac;
  y_inv_frac = 1 - y_frac;
  data_px[0] = (bl_val[0] * x_inv_frac + br_val[0] * x_frac) * y_inv_frac + (tl_val[0] * x_inv_frac + tr_val[0] * x_frac) * y_frac;
  data_px[1] = (bl_val[1] * x_inv_frac + br_val[1] * x_frac) * y_inv_frac + (tl_val[1] * x_inv_frac + tr_val[1] * x_frac) * y_frac;
  data_px[2] = (bl_val[2] * x_inv_frac + br_val[2] * x_frac) * y_inv_frac + (tl_val[2] * x_inv_frac + tr_val[2] * x_frac) * y_frac;
  return;
}

__device__
bool BB::rotate(int rot, int rows, int cols){
  auto anchor = Point{(float)x,(float)y};
  auto p1 = rotate_point(anchor, Point{(float)w,(float) 0}, rot, h + 1);
  auto p2 = rotate_point(anchor, Point{(float)0,(float)0}, rot, h + 1);
  auto p3 = rotate_point(anchor, Point{(float)w,(float)h}, rot, h + 1);
  auto p4 = rotate_point(anchor, Point{(float)0, (float)h}, rot, h + 1);
  auto xl = min(p1.x, min(p2.x, min(p3.x, p4.x)));
  auto xr = max(p1.x, max(p2.x, max(p3.x, p4.x)));
  auto yt = min(p1.y, min(p2.y, min(p3.y, p4.y)));
  auto yb = max(p1.y, max(p2.y, max(p3.y, p4.y)));
  if (xl < 0 || xr >= cols || yb >= rows || yt < 0) {
    return false;
  }
  x = ceil(xl);
  y = floor(yb);
  w = floor(xr) - x;
  h = y - ceil(yt);
  return true;
}


void rotate_matrix(int rows, int cols, int rot, float* out){
  for(int i=0;i<rows;i++){
    for(int j=0;j<cols;j++){
      auto p = Point{(float)j,(float)i};
      auto p_rot = rotate_point(Point{(float)0, (float)0}, p, rot, rows);
      out[(i*cols + j)*2] = p_rot.x;
      out[(i*cols + j)*2 + 1] = p_rot.y;
    }
  }
}