#include "hip/hip_runtime.h"
#include "util.cuh"

__device__
Point rotate_point(Point anchor, Point grid_pos, int rot, int query_rows){
  if (rot == 0){
    return Point{anchor.x + grid_pos.x, anchor.y - (query_rows - 1) + grid_pos.y};
  }
  float angle = (float)rot * M_PI/180;
  float x = (float)grid_pos.x * cos(angle) - ((float)(query_rows - 1 - grid_pos.y) * sin(angle));
  float y = -((float)grid_pos.x * sin(angle)) - ((float)(query_rows - 1 - grid_pos.y) * cos(angle));
  return Point{anchor.x + x, anchor.y + y};
}

// rotate the bounding box by 45 degrees
__device__
void BB::rotate(int rot){
  auto anchor = Point{(float)x,(float)y};
  auto p1 = rotate_point(anchor, Point{(float)w,(float) 0}, rot, h + 1);
  auto p2 = rotate_point(anchor, Point{(float)0,(float)0}, rot, h + 1);
  auto p3 = rotate_point(anchor, Point{(float)w,(float)h}, rot, h + 1);
  auto p4 = rotate_point(anchor, Point{(float)0, (float)h}, rot, h + 1);
  auto xl = min(p1.x, min(p2.x, min(p3.x, p4.x)));
  auto xr = max(p1.x, max(p2.x, max(p3.x, p4.x)));
  auto yt = max(p1.y, max(p2.y, max(p3.y, p4.y)));
  auto yb = min(p1.y, min(p2.y, min(p3.y, p4.y)));
  x = ceil(xl);
  y = floor(yt);
  w = floor(xr) - x;
  h = y - ceil(yb);
}

__device__
BB BB::intersect(const BB& other) const {
  BB ret;
  ret.x = max(x, other.x);
  ret.y = min(y, other.y);
  ret.w = min(x+w, other.x+other.w) - ret.x;
  ret.h = ret.y - max(y - h, other.y - other.h);
  return ret;
}

__device__
int get_value(float* arr, int i, int j, int rows, int cols){
  if(i < 0 || i >= rows || j < 0 || j >= cols){
    return 0;
  }
  return arr[i*cols + j];
}
__device__
int get_value(int* arr, int i, int j, int k, int rows, int cols){
  if(i < 0 || i >= rows || j < 0 || j >= cols){
    return 0;
  }
  return arr[i * cols * 3 + j * 3 + k];
}

__device__
float get_prefix_sum(const BB& bb, int rows, int cols,  float* ps_mat){
  auto mat_bb = BB{0, rows - 1, cols-1, rows-1};
  auto intersect_bb = bb.intersect(mat_bb);
  float ret = 0;
  ret += get_value(ps_mat, intersect_bb.y, intersect_bb.x+intersect_bb.w, rows, cols);
  ret -= get_value(ps_mat, intersect_bb.y, intersect_bb.x - 1, rows, cols);
  ret -= get_value(ps_mat, intersect_bb.y-intersect_bb.h - 1, intersect_bb.x + intersect_bb.w, rows, cols);
  ret += get_value(ps_mat, intersect_bb.y - intersect_bb.h - 1, intersect_bb.x - 1, rows, cols);
  return ret;
}


__device__
float bilinear_interpolate(Point p, int ch, int rows, int cols, int* data){
  
  int xl = floor(p.x);
  int yl = floor(p.y);
  int xh = ceil(p.x);
  int yh = ceil(p.y);
  auto bl_val = get_value(data, yl, xl, ch, rows, cols);
  auto br_val = get_value(data, yl, xh, ch, rows, cols);
  auto tl_val = get_value(data, yh, xl, ch, rows, cols);
  auto tr_val = get_value(data, yh, xh, ch, rows, cols);
  float x_frac, y_frac, x_inv_frac, y_inv_frac;
  if(xl == xh) {
    if (yl == yh) {
      return bl_val;
    }
    y_frac = (p.y - yl)/(yh - yl);
    y_inv_frac = 1 - y_frac;
    return (bl_val * y_inv_frac + tl_val * y_frac);
  }
  if(yl == yh) {
    x_frac = (p.x - xl)/(xh - xl);
    x_inv_frac = 1 - x_frac;
    return (bl_val * x_inv_frac + br_val * x_frac);
  }
  x_frac = (p.x - xl)/(xh - xl);
  y_frac = (p.y - yl)/(yh - yl);
  
  x_inv_frac = 1 - x_frac;
  y_inv_frac = 1 - y_frac;
  
  float ret = (bl_val * x_inv_frac + br_val * x_frac) * y_inv_frac + (tl_val * x_inv_frac + tr_val * x_frac) * y_frac;
  return ret;
}

