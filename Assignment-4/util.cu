#include "hip/hip_runtime.h"
#include "util.cuh"

__device__
Point rotate_point(Point anchor, Point grid_pos, int rot){
  if (rot == 0){
    return Point{grid_pos.x + anchor.x, grid_pos.y + anchor.y};
  }
  float angle = (float)rot * M_PI/180;
  float x = (float)grid_pos.x * cos(angle) - (float)grid_pos.y * sin(angle);
  float y = (float)grid_pos.x * sin(angle) + (float)grid_pos.y * cos(angle);
  return Point{x + anchor.x, y + anchor.y};
}

// rotate the bounding box by 45 degrees
__device__
void BB::rotate(int rot){
  //  switch(rot){
  //   case 45:
  //     float xr = x + (float)w/sqrt(2);
  //     float xl = x - (float)h/sqrt(2);
  //     float ht = y + (float)h/sqrt(2) + (float)w/sqrt(2);
  //     x = ceil(xl);
  //     y = y;
  //     w = (int)xr - x;
  //     h = (int)ht - y;

  //     break;
  //   case -45:
  //     float xr = x + (float)w/sqrt(2);
  //     float xl = x - (float)h/sqrt(2);
  //     float ht = y - (float)h/sqrt(2) - (float)w/sqrt(2);
  //     x = ceil(xl);
  //     y = y;
  //     w = (int)xr - x;
  //     h = ceil(ht) - y;
  //     break;
  //   default:
  //   break;
  // }
  // float angle = rot*M_PI/180;
  auto anchor = Point{(float)x,(float)y};
  auto p1 = rotate_point(anchor, Point{(float)x+w,(float)y}, rot);
  auto p2 = rotate_point(anchor, Point{(float)x+w,(float)y+h}, rot);
  auto p3 = rotate_point(anchor, Point{(float)x,(float)y+h}, rot);
  auto xl = min(p1.x, min(p2.x, p3.x));
  auto xr = max(p1.x, max(p2.x, p3.x));
  auto yt = max(p1.y, max(p2.y, p3.y));
  auto yb = min(p1.y, min(p2.y, p3.y));
  x = ceil(xl);
  y = ceil(yb);
  w = (int)xr - x;
  h = (int)yt - y;

}
__device__
int get_value(int* arr, int i, int j, int rows, int cols){
  if(i < 0 || i >= rows || j < 0 || j >= cols){
    return 255;
  }
  return arr[i*cols + j];
}

__device__
BB BB::intersect(const BB& other) const {
  BB ret;
  ret.x = max(x, other.x);
  ret.y = max(y, other.y);
  ret.w = min(x+w, other.x+other.w) - ret.x;
  ret.h = min(y+h, other.y+other.h) - ret.y;
  return ret;
}

__device__
float get_prefix_sum(const BB& bb, int rows, int cols,  int* ps_mat){
  auto mat_bb = BB{0,0,cols-1,rows-1};
  auto intersect_bb = bb.intersect(mat_bb);
  float ret = 0;
  ret += get_value(ps_mat, intersect_bb.y, intersect_bb.x, rows, cols);
  ret += get_value(ps_mat, intersect_bb.y+intersect_bb.h, intersect_bb.x+intersect_bb.w, rows, cols);
  ret -= get_value(ps_mat, intersect_bb.y+intersect_bb.h, intersect_bb.x, rows, cols);
  ret -= get_value(ps_mat, intersect_bb.y, intersect_bb.x+intersect_bb.w, rows, cols);
  auto remaining_pixels = bb.w*bb.h - intersect_bb.w * intersect_bb.h;
  ret += remaining_pixels*255;
  return ret;

}


__device__
float bilinear_interpolate(Point p, int rows, int cols, int* data){
  
  int xl = floor(p.x);
  int yl = floor(p.y);
  int xh = ceil(p.x);
  int yh = ceil(p.y);
  float x_frac = (p.x - xl)/(xh - xl);
  float y_frac = (p.y - yl)/(yh - yl);
  float x_inv_frac = 1 - x_frac;
  float y_inv_frac = 1 - y_frac;
  auto bl_val = get_value(data, yl, xl, rows, cols);
  auto br_val = get_value(data, yl, xh, rows, cols);
  auto tl_val = get_value(data, yh, xl, rows, cols);
  auto tr_val = get_value(data, yh, xh, rows, cols);
  float ret = (bl_val * x_inv_frac + br_val * x_frac) * y_inv_frac + (tl_val * x_inv_frac + tr_val * x_frac) * y_frac;
  return ret;
}

