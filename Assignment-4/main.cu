#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits>
#include <algorithm>
#include "img.hpp"
#include "util.cuh"


__global__
void find(float th1_cu, float th2_cu, int query_rows, int query_cols, int thread_count, int* data_cu, int* query_cu, float* prefixsum_cu, float graysum_cu, float* result_cu) {
    extern __shared__ float rmsd[];
    int row, col, rot, angle;
    row = blockIdx.x;
    col = blockIdx.y; 
    rot = blockIdx.z;
    if (rot == 0) {
        angle = 0;
    } else if (rot == 1) {
        angle = 45;
    } else {
        angle = -45;
    }
    // printf("Row: %d, col: %d, rot: %d\n", row, col, rot);
    // auto bb = BB{col, row, query_cols, query_rows };
    // bb.rotate(angle);
    // auto ps = get_prefix_sum(bb, blockDim.x, blockDim.y, prefixsum_cu);
    // if( abs(ps - graysum_cu) > th2_cu ){
    //     return;
    // }

    int tid = threadIdx.x;
    int chunk_sz = max(query_rows*query_cols/thread_count, 1);
    int start = tid*chunk_sz;
    int end = min(start+chunk_sz, query_rows*query_cols);
    rmsd[tid] = 0;
    
    for(auto query_idx = start; query_idx < end; query_idx++){
        int curr_col = ((query_idx / 3) % query_cols);
        int curr_row = ((query_idx / 3) / query_cols);
        
        for(auto ch=0; ch < 3; ch++){
            // filter
            
            //calculate rmsd
            float data_px, query_px;
            query_px = query_cu[query_idx + ch];
            
            auto rotated_point = rotate_point(Point{(float) col, (float) row}, Point{(float) curr_col, (float) curr_row}, angle);
            if (angle != 0) {
                data_px = bilinear_interpolate(rotated_point, blockDim.x, blockDim.y, data_cu);
            } else {
                data_px = get_value(data_cu, rotated_point.y, rotated_point.x, blockDim.x, blockDim.y);
            }
            rmsd[tid] += (data_px - query_px) * (data_px - query_px);
        }
        
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        float rmsd_val = 0;
        for (int i = 0; i < thread_count; i ++) {
            rmsd_val += rmsd[i];
        }
        result_cu[row * blockDim.y * blockDim.z + col * blockDim.z + rot] = sqrt(rmsd_val / (query_rows*query_cols*3));
    }
}

struct comparePairMax {
    bool operator() (const P<int, float>& a, const P<int, float>& b) {
        if (a.second < b.second) {
            return true;
        } else if (a.second == b.second) {
            return a.first < b.first;
        } else {
            return false;
        }
    }
};

int main(int argc, char** argv) {
    std::string data_img, query_img;
    float th1, th2;
    int n;
    data_img = argv[1];
    query_img = argv[2];
    th1 = std::stof(argv[3]);
    th2 = std::stof(argv[4]);
    n = std::stoi(argv[5]);
    auto data_mat = imread(data_img);
    auto query_mat = imread(query_img);
    T<int, int, int> data_sz = data_mat->shape(), query_sz = query_mat->shape();
    std::cout << "Data Image: " << std::get<0>(data_sz) << " " << std::get<1>(data_sz) << " " << std::get<2>(data_sz) << std::endl;
    std::cout << "Query Image: " << std::get<0>(query_sz) << " " << std::get<1>(query_sz) << " " << std::get<2>(query_sz) << std::endl;

    int data_mem = std::get<0>(data_sz) * std::get<1>(data_sz) * std::get<2>(data_sz);
    int query_mem = std::get<0>(query_sz) * std::get<1>(query_sz) * std::get<2>(query_sz);

    auto gray_data_mat = rbg2gray(data_mat.get());
    auto prefixsum_mat = prefixsum(gray_data_mat.get());
    auto graysum_val = graysum(query_mat.get());

    V<float> result_arr(data_mem, std::numeric_limits<float>::max());

    int *data_cu, *query_cu;
    float *prefixsum_cu, *result_cu;
    hipMalloc(&data_cu, data_mem * sizeof(int)); 
    hipMalloc(&query_cu, query_mem * sizeof(int));
    hipMalloc(&prefixsum_cu, (data_mem / std::get<2>(data_sz)) * sizeof(float));
    hipMalloc(&result_cu, data_mem * sizeof(float));

    hipMemcpy(data_cu, data_mat->get(), data_mem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(query_cu, query_mat->get(), query_mem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(prefixsum_cu, prefixsum_mat->get(), (data_mem / std::get<2>(data_sz)) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(result_cu, result_arr.data(), data_mem * sizeof(int), hipMemcpyHostToDevice);

    dim3 block_dim = dim3(std::get<0>(data_sz), std::get<1>(data_sz), std::get<2>(data_sz));
    dim3 thread_dim = dim3(std::min(1024, std::get<0>(query_sz) * std::get<1>(query_sz)));

    // invoke kernel
    find<<<block_dim, thread_dim, thread_dim.x * sizeof(float)>>>(th1, th2, std::get<0>(query_sz), std::get<1>(query_sz), thread_dim.x, data_cu, query_cu, prefixsum_cu, graysum_val, result_cu);
    // hipDeviceSynchronize();
    hipMemcpy(result_arr.data(), result_cu, data_mem * sizeof(float), hipMemcpyDeviceToHost);
    
    auto err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
    
    std::priority_queue<P<int, float>, V<P<int, float>>, comparePairMax> result_que;
    for (int i = 0; i < result_arr.size(); i ++) {
        if (result_arr[i] >= th1) {
            continue;
        }
        if (result_que.size() < n) {
            result_que.push(P<int, float>(i, result_arr[i]));
        } else {
            if (result_arr[i] >= result_que.top().second) {
                continue;
            } else {
                result_que.push(P<int, float>(i, result_arr[i]));
                result_que.pop();
            }
        }
    }
    
    V<T<int, int, int>> output(n);
    int index = result_que.size() - 1, output_sz = index + 1;
    std::ofstream fout("output.txt", std::ios::out);
    while (!result_que.empty()) {
        P<int, float> temp = result_que.top();
        result_que.pop();
        int row, col, rot;
        row = temp.first / (std::get<1>(data_sz) * std::get<2>(data_sz));
        col = (temp.first / (std::get<2>(data_sz))) % std::get<1>(data_sz);
        rot = temp.first % std::get<2>(data_sz);
        if (rot == 1) {
            rot = 45;
        } else if (rot == 2) {
            rot = -45;
        }
        output[index] = T<int, int, int>(row, col, rot);
        index --;
    }
    for (int i = 0; i < output_sz; i ++) {
        fout << std::get<0>(output[i]) << " " << std::get<1>(output[i]) << " " << std::get<2>(output[i]) << std::endl;
    }
    fout.close();
    hipFree(result_cu);
    hipFree(prefixsum_cu);
    hipFree(query_cu);
    hipFree(data_cu);
}
