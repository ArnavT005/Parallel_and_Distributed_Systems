#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits>
#include <algorithm>
#include "img.hpp"

__global__
void find(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char** argv)
{
    std::string data_img, query_img;
    double th1, th2;
    int n;
    data_img = argv[1];
    query_img = argv[2];
    th1 = std::stod(argv[3]);
    th2 = std::stod(argv[4]);
    n = std::stoi(argv[5]);
    auto data_mat = imread(data_img);
    auto query_mat = imread(query_img);
    T<int, int, int> data_sz = data_mat->shape(), query_sz = query_mat->shape();
    std::cout << "Data Image: " << std::get<0>(data_sz) << " " << std::get<1>(data_sz) << " " << std::get<2>(data_sz) << std::endl;
    std::cout << "Query Image: " << std::get<0>(query_sz) << " " << std::get<1>(query_sz) << " " << std::get<2>(query_sz) << std::endl;

    int data_mem = std::get<0>(data_sz) * std::get<1>(data_sz) * std::get<2>(data_sz);
    int query_mem = std::get<0>(query_sz) * std::get<1>(query_sz) * std::get<2>(query_sz);

    auto gray_data_mat = rbg2gray(data_mat.get());
    auto prefixsum_mat = prefixsum(gray_data_mat.get());
    auto graysum_val = graysum(query_mat.get());

    V<float> result_arr(data_mem, std::numeric_limits<float>::max());

    int *data_cu, *query_cu;
    float *prefixsum_cu, *graysum_cu, *result_cu;
    hipMalloc(&data_cu, data_mem * sizeof(int)); 
    hipMalloc(&query_cu, query_mem * sizeof(int)); 
    hipMalloc(&prefixsum_cu, (data_mem / std::get<2>(data_sz)) * sizeof(float));
    hipMalloc(&graysum_cu, sizeof(float));
    hipMalloc(&result_cu, data_mem * sizeof(float));

    hipMemcpy(data_cu, data_mat->get(), data_mem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(query_cu, query_mat->get(), query_mem * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(prefixsum_cu, prefixsum_mat->get(), (data_mem / std::get<2>(data_sz)) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(graysum_cu, &graysum_val, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(result_cu, result_arr.data(), data_mem * sizeof(float), hipMemcpyHostToDevice);

    dim3 block_dim = dim3(data_mem / std::get<2>(data_sz), 3);
    dim3 thread_dim = dim3(std::get<0>(query_sz), std::get<1>(query_sz), std::get<2>(query_sz));
    

    hipMemcpy(result_arr.data(), result_cu, data_mem * sizeof(float), hipMemcpyDeviceToHost);
    

    hipFree(graysum_cu);
    hipFree(prefixsum_cu);
    hipFree(query_cu);
    hipFree(data_cu);
}
